#include "hip/hip_runtime.h"
#include "GPUExecutor.hpp"
#include <hip/hip_runtime.h>

__global__ void vectorMultiply(float* d_data, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size) {
        d_data[idx] *= 2.0f;  // Example: Double each value
    }
}

GPUExecutor::GPUExecutor() {}

// Execute GPU-based computations
void GPUExecutor::executeOnGPU(std::vector<float>& data) {
    float* d_data;
    size_t size = data.size() * sizeof(float);

    hipMalloc(&d_data, size);
    hipMemcpy(d_data, data.data(), size, hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (data.size() + blockSize - 1) / blockSize;
    vectorMultiply<<<numBlocks, blockSize>>>(d_data, data.size());

    hipMemcpy(data.data(), d_data, size, hipMemcpyDeviceToHost);
    hipFree(d_data);

    std::cout << "GPU Execution: Data processed successfully.\n";
}
